// Code Adapted from: https://www.math.wustl.edu/~victor/mfmm/fourier/fft.c & https://www.geeksforgeeks.org/discrete-fourier-transform-and-its-inverse-using-c/

#include <assert.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define q 15
#define N (int)pow(2,q)
#define BLOCK_SIZE 256

typedef struct {
	double Re;
	double Im;
} complex;

#define PI	3.14159265358979323846264338327950288

void print_vector(const char* title, complex* x, int n) {
	int i;
	printf("%s (dim=%d):", title, n);
	for (i = 0; i < n; i++) printf(" %5.2f,%5.2f ", x[i].Re, x[i].Im);
	putchar('\n');
	return;
}

__global__ void dft_kernel(complex* v, int n, complex* tmp) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        tmp[i].Re = 0.0;
        tmp[i].Im = 0.0;
        for (int j = 0; j < n; j++) {
            tmp[i].Re += v[j].Re * cos(2 * M_PI * i * j / n) + 
				v[j].Im * sin(2 * M_PI * i * j / n);
            tmp[i].Im += -v[j].Re * sin(2 * M_PI * i * j / n) + 
				v[j].Im * cos(2 * M_PI * i * j / n);
        }
    }
}

void cudft(complex* v, int n, complex* tmp) {
    complex* dev_v;
    complex* dev_tmp;

    hipMalloc((void**)&dev_v, n * sizeof(complex));
    hipMalloc((void**)&dev_tmp, n * sizeof(complex));

    hipMemcpy(dev_v, v, n * sizeof(complex), hipMemcpyHostToDevice);

    int num_blocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    printf("Number of blocks used: %d\n", num_blocks);
    dft_kernel<<<num_blocks, BLOCK_SIZE>>>(dev_v, n, dev_tmp);

    hipMemcpy(tmp, dev_tmp, n * sizeof(complex), hipMemcpyDeviceToHost);

    hipFree(dev_v);
    hipFree(dev_tmp);

    for (int i = 0; i < n; i++) {
        v[i].Re = tmp[i].Re;
        v[i].Im = tmp[i].Im;
    }
}

void dft(complex* v, int n, complex* tmp) {
	for (int i = 0; i < n; i++) {
		tmp[i].Re = 0.0;
		tmp[i].Im = 0.0;
		for (int j = 0; j < n; j++) {
			tmp[i].Re = (tmp[i].Re + v[j].Re * cos(2 * PI * i * j / n));
			tmp[i].Im = (tmp[i].Im - v[j].Re * sin(2 * PI * i * j / n));
		}
	}
	for (int i = 0; i < n; i++) {
		v[i].Re = tmp[i].Re;
		v[i].Im = tmp[i].Im;
	}
	return;
}

void fillInput(complex* v, int n) {
	int sampleNum = n;
	double length = 10.0;
	double sample;

	for (int i = 0; i < sampleNum; i++) {
		double x = i * (length / sampleNum);
		sample = sin(20 * x) + cos(200 * x);
		v[i].Re = (double)(sample);
		v[i].Im = 0.0;
	}

	if (sampleNum < 17) {
		for (int i = 0; i < sampleNum; i++) {
			printf("%.2f ", v[i].Re);
		}
		printf("\n");
	}
}

int main() {
	// Allocate working and scratch arrays for complex numbers
	complex* v;
	v = (complex*)malloc(sizeof(complex) * N);
	complex* scratch;
	scratch = (complex*)malloc(sizeof(complex) * N);
	if (v == NULL || scratch == NULL)
		exit(-1);

	// Fill working array with complex numbers and display
	fillInput(v, N);
	if (N < 17)
		print_vector("Input", v, N);

	// Perform the dft on the input
	clock_t start, end;
	double cpu_time;
	start = clock();
	cudft(v, N, scratch);
	end = clock();
	if (N < 17)
		print_vector("DFT", v, N);
	cpu_time = ((double)(end - start)) / CLOCKS_PER_SEC;
	printf("DFT: Time taken is %.3f\n", cpu_time);

	// Deallocate arrays	
	free(v);
	free(scratch);

  return 0;
}